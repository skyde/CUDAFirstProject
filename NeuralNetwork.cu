#pragma once
#include <stdio.h>
#include <iostream>
#include <array>
#include "hip/hip_runtime_api.h"
#include <stdlib.h>

#include "SharedData.cu"
#include "Layer.cu"
#include "Element.cu"
#include "Node.cu"
//#include "NeuralNetworkDevice.cu"
using namespace std;

// Total Threads
#define N 2 // Nodes per layer
// Block Size
#define M 1 // 512

#define LAYERS 2

#define PRINT_DERIVATIVE true

//template <class T>
class NeuralNetwork
{
public:
	NeuralNetwork()
	{
//		array<SharedData<Node>*, LAYERS> layers;
//		array<SharedData<Element>*, LAYERS - 1> weights;

		for(int i = 0; i < layers.size(); i++)
		{
			layers[i] = new SharedData<Node>(N);

			if(i != 0)
			{
				int length = layers[i - 1]->Length * layers[i]->Length;

				weights[i - 1] = new SharedData<Element>(length);

				cout << "weights " << i - 1 << ", l = " << length << "\n";
			}
		}
	}

	array<SharedData<Node>*, LAYERS> layers;
	array<SharedData<Element>*, LAYERS - 1> weights;

//	void randomValues(double* a, int n);
//	void randomValues(Node* a, int n);
//	void randomValues(Element* a, int n);

//	void Forward()
//	{
//		ForwardPass<<<N / M, M>>>(
//				layers[0]->DeviceData,
//				layers[0]->Length,
//				weights[0]->DeviceData,
//				layers[1]->DeviceData,
//				layers[1]->Length);
//	}
//
//	void Backward()
//	{
//		BackwardPass<<<N / M, M>>>(
//				layers[0]->DeviceData,
//				layers[0]->Length,
//				weights[0]->DeviceData,
//				layers[1]->DeviceData,
//				layers[1]->Length);
//	}

	void CopyToDevice()
	{
		for(int i = 0; i < layers.size(); ++i)
		{
			layers[i]->CopyToDevice();
		}

		for(int i = 0; i < weights.size(); ++i)
		{
			weights[i]->CopyToDevice();
		}
	}

	void CopyToHost()
	{
		for(int i = 0; i < layers.size(); ++i)
		{
			layers[i]->CopyToHost();
		}

		for(int i = 0; i < weights.size(); ++i)
		{
			weights[i]->CopyToHost();
		}
	}

	void Print()
	{
		for(int y = 0; y < N; y++)
		{
			for(int x = 0; x < LAYERS; x++)
			{
				((Node)layers[x]->HostData[y]).Print();
				cout << " ";

				int nextLayer = x + 1;
				if(nextLayer < LAYERS)
				{
					int block = layers[nextLayer]->Length;
					for(int w = 0; w < block; w++)
					{
						int index = y * block + w;

						cout << y << "->" << w;

						((Element)weights[x]->HostData[index]).Print();

						cout << " ";
					}
				}
			}

			cout << "\n";
		}
	}

	inline double randomValue()
	{
		return 0.5 + (double)rand() / RAND_MAX;
	}

	void randomValues(double* a, int n)
	{
		int i;
		for (i = 0; i < n; ++i)
		{
			a[i] = randomValue();
		}
	}

	void randomValues(Node* a, int n)
	{
		int i;
		for (i = 0; i < n; ++i)
		{
			a[i].Self.Value = randomValue();
			a[i].Bias.Value = randomValue();
		}
	}

	void randomValues(Element* a, int n)
	{
		int i;
		for (i = 0; i < n; ++i)
		{
			a[i].Value = randomValue();
		}
	}

	virtual ~NeuralNetwork()
	{
		for(int i = 0; i < layers.size(); i++)
		{
			delete layers[i];
		}

		for(int i = 0; i < weights.size(); i++)
		{
			delete weights[i];
		}
	}
};
