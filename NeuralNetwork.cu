#pragma once
#include <stdio.h>
#include <iostream>
#include <array>
#include "hip/hip_runtime_api.h"
#include <stdlib.h>

#include "SharedData.cu"
#include "Layer.cu"
#include "Element.cu"
#include "Node.cu"
//#include "NeuralNetworkDevice.cu"
using namespace std;

// Total Threads
#define N 2 // Nodes per layer
// Block Size
#define M 1 // 512

#define LAYERS 2

#define PRINT_DERIVATIVE true

//template <class T>
class NeuralNetwork
{
public:
	NeuralNetwork()
	{
//		array<SharedData<Node>*, LAYERS> layers;
//		array<SharedData<Element>*, LAYERS - 1> weights;

		for(int i = 0; i < Layers.size(); i++)
		{
			Layers[i] = new SharedData<Node>(N);

			if(i != 0)
			{
				int length = Layers[i - 1]->Length * Layers[i]->Length;

				Weights[i - 1] = new SharedData<Element>(length);

				cout << "weights " << i - 1 << ", l = " << length << "\n";
			}
		}
	}

	array<SharedData<Node>*, LAYERS> Layers;
	array<SharedData<Element>*, LAYERS - 1> Weights;

//	void randomValues(double* a, int n);
//	void randomValues(Node* a, int n);
//	void randomValues(Element* a, int n);

//	void Forward()
//	{
//		ForwardPass<<<N / M, M>>>(
//				layers[0]->DeviceData,
//				layers[0]->Length,
//				weights[0]->DeviceData,
//				layers[1]->DeviceData,
//				layers[1]->Length);
//	}
//
//	void Backward()
//	{
//		BackwardPass<<<N / M, M>>>(
//				layers[0]->DeviceData,
//				layers[0]->Length,
//				weights[0]->DeviceData,
//				layers[1]->DeviceData,
//				layers[1]->Length);
//	}

	void CopyToDevice()
	{
		for(int i = 0; i < Layers.size(); ++i)
		{
			Layers[i]->CopyToDevice();
		}

		for(int i = 0; i < Weights.size(); ++i)
		{
			Weights[i]->CopyToDevice();
		}
	}

	void CopyToHost()
	{
		for(int i = 0; i < Layers.size(); ++i)
		{
			Layers[i]->CopyToHost();
		}

		for(int i = 0; i < Weights.size(); ++i)
		{
			Weights[i]->CopyToHost();
		}
	}

	void Print()
	{
		for(int y = 0; y < N; y++)
		{
			for(int x = 0; x < LAYERS; x++)
			{
				((Node)Layers[x]->HostData[y]).Print();
				cout << " ";

				int nextLayer = x + 1;
				if(nextLayer < LAYERS)
				{
					int block = Layers[nextLayer]->Length;
					for(int w = 0; w < block; w++)
					{
						int index = y * block + w;

						cout << y << "->" << w;

						((Element)Weights[x]->HostData[index]).Print();

						cout << " ";
					}
				}
			}

			cout << "\n";
		}
	}

	inline double randomValue()
	{
		return 0.5 + (double)rand() / RAND_MAX;
	}

	void randomValues(double* a, int n)
	{
		int i;
		for (i = 0; i < n; ++i)
		{
			a[i] = randomValue();
		}
	}

	void randomValues(Node* a, int n)
	{
		int i;
		for (i = 0; i < n; ++i)
		{
			a[i].Self.Value = randomValue();
			a[i].Bias.Value = randomValue();
		}
	}

	void randomValues(Element* a, int n)
	{
		int i;
		for (i = 0; i < n; ++i)
		{
			a[i].Value = randomValue();
		}
	}

	virtual ~NeuralNetwork()
	{
		for(int i = 0; i < Layers.size(); i++)
		{
			delete Layers[i];
		}

		for(int i = 0; i < Weights.size(); i++)
		{
			delete Weights[i];
		}
	}
};
