#include "hip/hip_runtime.h"
#pragma once
#include <stdio.h>
#include <iostream>
#include <array>
#include "hip/hip_runtime_api.h"
#include <stdlib.h>
#include <random>

#include "SharedData.cu"
#include "Layer.cu"
#include "Element.cu"
#include "Node.cu"
#include "Globals.h"
using namespace std;

class NeuralNetwork
{
public:
	NeuralNetwork()
	{
		normal_distribution<double> distribution(0, 1.0);

		for(int i = 0; i < Layers.size(); i++)
		{
			cout << "Nodes in Layer " << i << " " << NodesInLayer(i) << "\n";
			Layers[i] = new SharedData<Node>(NodesInLayer(i));

			if(i != 0)
			{
				int length = Layers[i - 1]->Length * Layers[i]->Length;

				Weights[i - 1] = new SharedData<Element>(length);

				randomValues(
						Weights[i - 1]->HostData,
						Weights[i - 1]->Length,
						i);

//				cout << "weights " << i - 1 << ", l = " << length << "\n";
			}

			if(i > 0 && i < Layers.size() - 1)
			{
				randomValues(
						Layers[i]->HostData,
						Layers[i]->Length,
						i);
			}
		}
	}

	array<SharedData<Node>*, LAYERS> Layers;
	array<SharedData<Element>*, LAYERS - 1> Weights;

	void CopyToDevice()
	{
		for(int i = 0; i < Layers.size(); ++i)
		{
			Layers[i]->CopyToDevice();
		}

		for(int i = 0; i < Weights.size(); ++i)
		{
			Weights[i]->CopyToDevice();
		}
	}

	void CopyToHost()
	{
		for(int i = 0; i < Layers.size(); ++i)
		{
			Layers[i]->CopyToHost();
		}

		for(int i = 0; i < Weights.size(); ++i)
		{
			Weights[i]->CopyToHost();
		}
	}

	double CaculateError(double* targets, bool print = false)
	{
		SharedData<Node>* outputs = Layers[Layers.size() - 1];

		double value = 0;

		for(int i = 0; i < outputs->Length; i++)
		{
			double target = outputs->HostData[i].Self.Value;

			double diff = abs(targets[i] - target);

			if(print)
			{
				cout << "(" << diff << ")";
			}

			value += diff * diff;
		}

		return value;
	}

	void PrintVerbose()
	{
		for(int y = 0; y < NodesInLayer(0); y++)
		{
			for(int x = 0; x < LAYERS; x++)
			{
				if(x >= NodesInLayer(y))
				{
					continue;
				}

				((Node)Layers[x]->HostData[y]).Print();
				cout << " ";

				int nextLayer = x + 1;
				if(nextLayer < LAYERS)
				{
					int block = Layers[nextLayer]->Length;
					for(int w = 0; w < block; w++)
					{
						int index = y * block + w;

						cout << y << "->" << w;

						((Element)Weights[x]->HostData[index]).Print();

						cout << " ";
					}
				}
			}

			cout << "\n";
		}
	}

	default_random_engine generator;
	normal_distribution<double> distribution;

	inline double randomValue(int layer)
	{
		return (distribution(generator) / NodesInLayer(layer)) * 0.001;
	}

//	void randomValues(double* a, int n)
//	{
//		int i;
//		for (i = 0; i < n; ++i)
//		{
//			a[i] = randomValue();
//		}
//	}

	void randomValues(Node* a, int n, int layer)
	{
		int i;
		for (i = 0; i < n; ++i)
		{
//			a[i].Self.Value = randomValue();
			a[i].Bias.Value = randomValue(layer);
		}
	}

	void randomValues(Element* a, int n, int layer)
	{
		int i;
		for (i = 0; i < n; ++i)
		{
			a[i].Value = randomValue(layer);
		}
	}

	virtual ~NeuralNetwork()
	{
		for(int i = 0; i < Layers.size(); i++)
		{
			delete Layers[i];
		}

		for(int i = 0; i < Weights.size(); i++)
		{
			delete Weights[i];
		}
	}
};
