#pragma once
#include <stdio.h>
#include <iostream>
#include <array>
#include "hip/hip_runtime_api.h"
#include <stdlib.h>
#include <random>

#include "SharedData.cu"
#include "Layer.cu"
#include "Element.cu"
#include "Node.cu"
#include "Globals.h"
//#include "NeuralNetworkDevice.cu"
using namespace std;


//template <class T>
class NeuralNetwork
{
public:
	NeuralNetwork()
	{
		normal_distribution<double> distribution(0, 1.0);
//		array<SharedData<Node>*, LAYERS> layers;
//		array<SharedData<Element>*, LAYERS - 1> weights;

		for(int i = 0; i < Layers.size(); i++)
		{
			Layers[i] = new SharedData<Node>(N);

			if(i != 0)
			{
				int length = Layers[i - 1]->Length * Layers[i]->Length;

				Weights[i - 1] = new SharedData<Element>(length);

				randomValues(
						Weights[i - 1]->HostData,
						Weights[i - 1]->Length);

//				cout << "weights " << i - 1 << ", l = " << length << "\n";
			}

			randomValues(
					Layers[i]->HostData,
					Layers[i]->Length);
		}
	}

	array<SharedData<Node>*, LAYERS> Layers;
	array<SharedData<Element>*, LAYERS - 1> Weights;

//	void randomValues(double* a, int n);
//	void randomValues(Node* a, int n);
//	void randomValues(Element* a, int n);

//	void Forward()
//	{
//		ForwardPass<<<N / M, M>>>(
//				layers[0]->DeviceData,
//				layers[0]->Length,
//				weights[0]->DeviceData,
//				layers[1]->DeviceData,
//				layers[1]->Length);
//	}
//
//	void Backward()
//	{
//		BackwardPass<<<N / M, M>>>(
//				layers[0]->DeviceData,
//				layers[0]->Length,
//				weights[0]->DeviceData,
//				layers[1]->DeviceData,
//				layers[1]->Length);
//	}

	void CopyToDevice()
	{
		for(int i = 0; i < Layers.size(); ++i)
		{
			Layers[i]->CopyToDevice();
		}

		for(int i = 0; i < Weights.size(); ++i)
		{
			Weights[i]->CopyToDevice();
		}
	}

	void CopyToHost()
	{
		for(int i = 0; i < Layers.size(); ++i)
		{
			Layers[i]->CopyToHost();
		}

		for(int i = 0; i < Weights.size(); ++i)
		{
			Weights[i]->CopyToHost();
		}
	}

	void Print()
	{
		for(int y = 0; y < N; y++)
		{
			for(int x = 0; x < LAYERS; x++)
			{
				((Node)Layers[x]->HostData[y]).Print();
				cout << " ";

				int nextLayer = x + 1;
				if(nextLayer < LAYERS)
				{
					int block = Layers[nextLayer]->Length;
					for(int w = 0; w < block; w++)
					{
						int index = y * block + w;

						cout << y << "->" << w;

						((Element)Weights[x]->HostData[index]).Print();

						cout << " ";
					}
				}
			}

			cout << "\n";
		}
	}

	default_random_engine generator;
	normal_distribution<double> distribution;

	inline double randomValue()
	{
		return (distribution(generator) / N) * 0.01;
	}

//	void randomValues(double* a, int n)
//	{
//		int i;
//		for (i = 0; i < n; ++i)
//		{
//			a[i] = randomValue();
//		}
//	}

	void randomValues(Node* a, int n)
	{
		int i;
		for (i = 0; i < n; ++i)
		{
//			a[i].Self.Value = randomValue();
			a[i].Bias.Value = randomValue();
		}
	}

	void randomValues(Element* a, int n)
	{
		int i;
		for (i = 0; i < n; ++i)
		{
			a[i].Value = randomValue();
		}
	}

	virtual ~NeuralNetwork()
	{
		for(int i = 0; i < Layers.size(); i++)
		{
			delete Layers[i];
		}

		for(int i = 0; i < Weights.size(); i++)
		{
			delete Weights[i];
		}
	}
};
