#include <iostream>
using namespace std;
#include "hip/hip_runtime_api.h"

//template <class T>
class SharedData
{
public:
	SharedData(int length)
	{
		Length = length;
		TotalBytes = length * sizeof(double);

		HostData = (double *) malloc(TotalBytes);
		checkCudaErrors(hipMalloc((void **)& DeviceData, TotalBytes));

		cout << "construct" << "\n";
	}

	virtual ~SharedData()
	{
	}

	void Dispose()
	{
		free(HostData);
		hipFree(DeviceData);

		cout << "deconstruct" << "\n";
	}

	void CopyToDevice()
	{
		checkCudaErrors(hipMemcpy(DeviceData, HostData, TotalBytes, hipMemcpyHostToDevice));
	}

	void CopyToHost()
	{
		checkCudaErrors(hipMemcpy(HostData, DeviceData, TotalBytes, hipMemcpyDeviceToHost));
	}

	int TotalBytes;
	int Length;

	double *HostData;
	double *DeviceData;
};
