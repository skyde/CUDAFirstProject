#pragma once

#include <iostream>
using namespace std;
#include "hip/hip_runtime_api.h"

template <class T>
class SharedData
{
public:
	SharedData(int length)
	{
		Length = length;
		TotalBytes = length * sizeof(T);

		HostData = (T *) calloc(1, TotalBytes);
		checkCudaErrors(hipMalloc((void **)& DeviceData, TotalBytes));

//		for(int i = 0; i < length; i++)
//		{
//			HostData[i] = new T();
//		}

//		cout << "construct" << "\n";
	}

	virtual ~SharedData()
	{
		free(HostData);
		hipFree(DeviceData);

//		cout << "deconstruct" << "\n";
	}

//	void Dispose()
//	{
//	}

	void CopyToDevice()
	{
		checkCudaErrors(hipMemcpy(DeviceData, HostData, TotalBytes, hipMemcpyHostToDevice));
	}

	void CopyToHost()
	{
		checkCudaErrors(hipMemcpy(HostData, DeviceData, TotalBytes, hipMemcpyDeviceToHost));
	}

	int TotalBytes;
	int Length;

	T *HostData;
	T *DeviceData;
};
