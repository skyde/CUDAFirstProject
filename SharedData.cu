#include <iostream>
using namespace std;
#include "hip/hip_runtime_api.h"

template <class T> class SharedData
{
public:
	SharedData(int length) : Length(length), TotalBytes(length * sizeof(T))
	{
		HostData = (T *) malloc(Length);
		checkCudaErrors(hipMalloc((void **)& DeviceData, TotalBytes));

		cout << "construct" << "\n";
	}

	virtual ~SharedData()
	{
	}

	void Dispose()
	{
		free(HostData);
		hipFree(DeviceData);

		cout << "deconstruct" << "\n";
	}

	void CopyToDevice()
	{
		checkCudaErrors(hipMemcpy(DeviceData, HostData, TotalBytes, hipMemcpyHostToDevice));
	}

	void CopyToHost()
	{
		checkCudaErrors(hipMemcpy(HostData, DeviceData, TotalBytes, hipMemcpyDeviceToHost));
	}

	const int TotalBytes;
	const int Length;

	T *HostData;
	T *DeviceData;
};
