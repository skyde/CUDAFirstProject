
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

template <class T> class SharedData
{
public:
	SharedData(int length) : Length(length), TotalBytes(length * sizeof(T))
	{
		HostData = (T *) malloc(Length);
		hipMalloc((void **)& DeviceData, TotalBytes);

//		cout << "construct" << "\n";
	}

	virtual ~SharedData()
	{
		free(HostData);
		hipFree(DeviceData);

//		cout << "deconstruct" << "\n";
	}

	const int TotalBytes;
	const int Length;

	T *HostData;
	T *DeviceData;
};
