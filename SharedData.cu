
#include <hip/hip_runtime.h>
/*
// * Layer.h
// *
// *  Created on: Apr 20, 2016
// *      Author: admin
// */
//
//#ifndef LAYER_H_
//#define LAYER_H_

template <class T> class SharedData
{
public:
	SharedData(int length) : Length(length), TotalBytes(length * sizeof(T))
	{
		HostData = (T *) malloc(Length);
		hipMalloc((void **)& DeviceData, TotalBytes);
	}

	virtual ~SharedData()
	{
		free(HostData);
		hipFree(DeviceData);
	}

	const int TotalBytes;
	const int Length;

	T *HostData;
	T *DeviceData;
};

//#endif /* LAYER_H_ */
