
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

template <class T> class SharedData
{
public:
	SharedData(int length) : Length(length), TotalBytes(length * sizeof(T))
	{
		HostData = (T *) malloc(Length);
		hipMalloc((void **)& DeviceData, TotalBytes);

		cout << "construct" << "\n";
	}

	virtual ~SharedData()
	{
		free(HostData);
		hipFree(DeviceData);

		cout << "deconstruct" << "\n";
	}

	void CopyToDevice()
	{
		hipMemcpy(DeviceData, HostData, TotalBytes, hipMemcpyHostToDevice);
	}

	void CopyToHost()
	{
		hipMemcpy(HostData, DeviceData, TotalBytes, hipMemcpyDeviceToHost);
	}

	const int TotalBytes;
	const int Length;

	T *HostData;
	T *DeviceData;
};
