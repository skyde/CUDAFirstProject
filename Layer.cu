
#include <hip/hip_runtime.h>
//#pragma once
//
//#include <iostream>
//using namespace std;
//#include "helper_cuda.h"
//#include "SharedData.cu"
//
//template <class T>
//class Layer
//{
//public:
//	Layer(int length)
//	{
//		Values = new SharedData<T>(length);
//		Biases = new SharedData<T>(length);
//		Derivatives = new SharedData<T>(length);
//	}
//
//	virtual ~Layer()
//	{
//	}
//
//	SharedData<T>* Values;
//	SharedData<T>* Biases;
//	SharedData<T>* Derivatives;
//
////	void Dispose()
////	{
////		free(HostData);
////		cudaFree(DeviceData);
////
////		cout << "deconstruct" << "\n";
////	}
//
////	void CopyToDevice()
////	{
////		checkCudaErrors(cudaMemcpy(DeviceData, HostData, TotalBytes, cudaMemcpyHostToDevice));
////	}
////
////	void CopyToHost()
////	{
////		checkCudaErrors(cudaMemcpy(HostData, DeviceData, TotalBytes, cudaMemcpyDeviceToHost));
////	}
//
////	int TotalBytes;
////	int Length;
////
////	T *HostData;
////	T *DeviceData;
//};
