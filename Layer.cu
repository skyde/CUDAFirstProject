#pragma once

#include <iostream>
using namespace std;
#include "hip/hip_runtime_api.h"
#include "SharedData.cu"

//template <class T>
class Layer
{
public:
	Layer(int length)
	{
		Values = new SharedData<double>(length);
		Biases = new SharedData<double>(length);
		Derivatives = new SharedData<double>(length);
	}

	virtual ~Layer()
	{
	}

	SharedData<double>* Values;
	SharedData<double>* Biases;
	SharedData<double>* Derivatives;

//	void Dispose()
//	{
//		free(HostData);
//		hipFree(DeviceData);
//
//		cout << "deconstruct" << "\n";
//	}

//	void CopyToDevice()
//	{
//		checkCudaErrors(hipMemcpy(DeviceData, HostData, TotalBytes, hipMemcpyHostToDevice));
//	}
//
//	void CopyToHost()
//	{
//		checkCudaErrors(hipMemcpy(HostData, DeviceData, TotalBytes, hipMemcpyDeviceToHost));
//	}

//	int TotalBytes;
//	int Length;
//
//	T *HostData;
//	T *DeviceData;
};
