
#include <hip/hip_runtime.h>
//#include <iostream>
//using namespace std;
//#include "helper_cuda.h"
//
//template <class T>
//class Layer
//{
//public:
//	SharedData(int length)
//	{
//		Length = length;
//		TotalBytes = length * sizeof(T);
//
//		HostData = (double *) malloc(TotalBytes);
//		checkCudaErrors(cudaMalloc((void **)& DeviceData, TotalBytes));
//
//		cout << "construct" << "\n";
//	}
//
//	virtual ~SharedData()
//	{
//	}
//
////	void Dispose()
////	{
////		free(HostData);
////		cudaFree(DeviceData);
////
////		cout << "deconstruct" << "\n";
////	}
//
////	void CopyToDevice()
////	{
////		checkCudaErrors(cudaMemcpy(DeviceData, HostData, TotalBytes, cudaMemcpyHostToDevice));
////	}
////
////	void CopyToHost()
////	{
////		checkCudaErrors(cudaMemcpy(HostData, DeviceData, TotalBytes, cudaMemcpyDeviceToHost));
////	}
//
//	int TotalBytes;
//	int Length;
//
//	T *HostData;
//	T *DeviceData;
//};
