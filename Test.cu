#include "hip/hip_runtime.h"

//#include "GL/glut.h"
// OpenGL Graphics includes
//#include <GL/glew.h>
//#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
//#include <GL/wglew.h>
//#endif
//#if defined(__APPLE__) || defined(__MACOSX)
//  #pragma clang diagnostic ignored "-Wdeprecated-declarations"
//  #include <GLUT/glut.h>
//  #ifndef glutCloseFunc
//  #define glutCloseFunc glutWMCloseFunc
//  #endif
//#else
//#include <GL/freeglut.h>
//#endif
//
//// CUDA runtime
//// CUDA utilities and system includes
//#include <hip/hip_runtime.h>
//#include <cuda_gl_interop.h>
//
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>
//#include <helper_cuda_gl.h>
//#include <rendercheck_gl.h>


#include <stdio.h>
#include <iostream>
#include "hip/hip_runtime_api.h"

#include "SharedData.cu"
using namespace std;

void randomValues(double* a, int n);
//void initGL(int *argc, char **argv);

// Total Threads
#define N 16 // 4096
// Block Size
#define M 2 // 512

#define RADIUS 1

class Layer
{
public:

};

//__global__ void add(int *a, int *b, int *c, int n)
//{
//	int index = threadIdx.x + blockIdx.x * blockDim.x;
//
//	if(index < n)
//	{
//		c[index] = a[index] + b[index];
//	}
//}

__global__ void ForwardPass(
		double* leftValues,
		double* weights, // left to right
		double* rightValues, // write target
		double* rightBiases)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	double output = leftValues[index];

	for(int i = 0; i < N; i++)
	{
		output *= weights[index * N + i];
	}

	rightValues[index] = tanh(output + rightBiases[index]);
}

__global__ void BackwardPass(
		double* leftValues,
		double* leftBiases,
		double* leftDerivatives,
		double* weights,
		double* rightValues,
		double* rightBiases,
		double* rightDerivatives)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	double output = leftValues[index];

	for(int i = 0; i < N; i++)
	{
		output *= weights[index * N + i];
	}

	rightValues[index] = tanh(output + rightBiases[index]);
}
//__global__ void BackwardPass(
//		double* left,
//		double* weights, // left to right
//		double* right,
//		double* biases)
//{
//	int index = threadIdx.x + blockIdx.x * blockDim.x;
//
//	double output = left[index];
//
//	for(int i = 0; i < N; i++)
//	{
//		output *= weights[index * N + i];
//	}
//
//	right[index] = tanh(output + biases[index]);
//}

int main(int argc, char **argv)
{
	printf ("N = %d \n", N);

	SharedData<double>* leftValues = new SharedData<double>(N);
	SharedData<double>* weights = new SharedData<double>(N * N);
	SharedData<double>* rightValues = new SharedData<double>(N);
	SharedData<double>* rightBiases = new SharedData<double>(N);

	randomValues(leftValues->HostData, leftValues->Length);
	randomValues(weights->HostData, weights->Length);
	randomValues(rightBiases->HostData, rightBiases->Length);

	cout << "Generated random values\n";

	leftValues->CopyToDevice();
	weights->CopyToDevice();
	rightBiases->CopyToDevice();

	cout << "Copy to device calls after initiated\n";

//	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
//	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

//	dim3 threadsPerBlock(16, 16);
//	dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);

	ForwardPass<<<N / M, M>>>(
			leftValues->DeviceData,
			weights->DeviceData,
			rightValues->DeviceData,
			rightBiases->DeviceData);

	cout << "RunPass initiated\n";

	rightValues->CopyToHost();

	cout << "CopyToHost initiated\n";

    hipDeviceSynchronize();

	cout << "hipDeviceSynchronize finished\n";

    getLastCudaError("Device kernel execution failed.\n");

    cout << "Execution finished, will print\n";

	for(int i = 0; i < rightValues->Length && i < 512; i++)
	{
		cout << leftValues->HostData[i] << " = " << rightValues->HostData[i] << "\n";
	}

    cout << "print finished\n";

	leftValues->Dispose();
	weights->Dispose();
	rightValues->Dispose();
	rightBiases->Dispose();

    cout << "dispose finished\n";

    hipDeviceReset();

    cout << "hipDeviceReset finished\n";

	delete leftValues;
	delete weights;
	delete rightValues;
	delete rightBiases;

    cout << "will exit\n";

//    exit(EXIT_SUCCESS);

	return 0;
}

void randomValues(double* a, int n)
{
	int i;
	for (i = 0; i < n; ++i)
	{
		a[i] = 0.5 + (double)rand() / RAND_MAX;

		cout << "random = " << a[i] << "\n";
	}
}
