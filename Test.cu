#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <array>
#include "hip/hip_runtime_api.h"

#include "SharedData.cu"
#include "Layer.cu"
using namespace std;

//void initGL(int *argc, char **argv);

// Total Threads
#define N 2 // Nodes per layer
// Block Size
#define M 1 // 512

#define LAYERS 2

#define PRINT_DERIVATIVE false

//#define RADIUS 1


//__global__ void add(int *a, int *b, int *c, int n)
//{
//	int index = threadIdx.x + blockIdx.x * blockDim.x;
//
//	if(index < n)
//	{
//		c[index] = a[index] + b[index];
//	}
//}

struct __align__(sizeof(double) * 2) Element
{
	Element() : Value(0), Derivative(0)
	{
//		cout << "Element ctor";
	}

    double Value, Derivative;

	public:
	void Print()
	{
		cout << "[" << Value;
#if PRINT_DERIVATIVE
		cout << " " << Derivative;
#endif
		cout << "]";
	}
};

struct __align__(sizeof(Element) * 2) Node
{
	Node() : Self(), Bias()
	{
//		cout << "Node ctor";
	}

	Element Self, Bias;

	public:
	void Print()
	{
		cout << "(" << Self.Value;
#if PRINT_DERIVATIVE
		cout << " " << Self.Derivative;
#endif
		cout << ")";

//		cout << "(" << Self.Value << " " << Self.Derivative << ")";
	}
};

void randomValues(double* a, int n);
void randomValues(Node* a, int n);
void randomValues(Element* a, int n);

__global__ void ForwardPass(
		Node* left,
		int leftLength,
		Element* weights, // left to right
		Node* right,
		int rightLength)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	double value = 0;//left[index].Self.Value;

	for(int i = 0; i < leftLength; i++)
	{
		int lookup = index + i * rightLength;

		value += left[i].Self.Value * weights[lookup].Value;
	}

	right[index].Self.Value = value;// + right[index].Bias.Value
}

__global__ void BackwardPass(
		Node* left,
		int leftLength,
		Element* weights,
		Node* right,
		int rightLength)
{
//	int index = threadIdx.x + blockIdx.x * blockDim.x;
//
//	double output = left[index].Self.Value;
//
//	for(int i = 0; i < N; i++)
//	{
//		output *= weights[index * N + i].Value;
//	}
//
//	right[index].Self.Value = tanh(output + right[index].Bias.Value);
}

int main(int argc, char **argv)
{
	printf ("N = %d \n", N);

	array<SharedData<Node>*, LAYERS> layers;
	array<SharedData<Element>*, LAYERS - 1> weights;

	for(int i = 0; i < layers.size(); i++)
	{
		layers[i] = new SharedData<Node>(N);

		if(i != 0)
		{
			int length = layers[i - 1]->Length * layers[i]->Length;

			weights[i - 1] = new SharedData<Element>(length);

			cout << "weights " << i - 1 << ", l = " << length << "\n";
		}
	}

	layers[0]->HostData[0].Self.Value = 1;
	layers[0]->HostData[1].Self.Value = 10;

//	layers[1]->HostData[0].Self.Value = 10;
//	layers[1]->HostData[1].Self.Value = 20;

	weights[0]->HostData[0].Value = 1;
	weights[0]->HostData[1].Value = 0.5;
	weights[0]->HostData[2].Value = 0;
	weights[0]->HostData[3].Value = 1;

	cout << "Generated random values\n";

	layers[0]->CopyToDevice();
	weights[0]->CopyToDevice();
	layers[1]->CopyToDevice();

	cout << "Copy to device calls after initiated\n";

	ForwardPass<<<N / M, M>>>(
			layers[0]->DeviceData,
			layers[0]->Length,
			weights[0]->DeviceData,
			layers[1]->DeviceData,
			layers[1]->Length);

    hipDeviceSynchronize();

	BackwardPass<<<N / M, M>>>(
			layers[0]->DeviceData,
			layers[0]->Length,
			weights[0]->DeviceData,
			layers[1]->DeviceData,
			layers[1]->Length);

	cout << "RunPass initiated\n";

	layers[1]->CopyToHost();

	cout << "CopyToHost initiated\n";

    hipDeviceSynchronize();

	cout << "hipDeviceSynchronize finished\n";

    getLastCudaError("Device kernel execution failed.\n");

    cout << "Execution finished, will print\n";

//    int numLayers = 2;
//    int nodesPerLayer = N;

	for(int y = 0; y < N; y++)
	{
		for(int x = 0; x < LAYERS; x++)
		{
			layers[x]->HostData[y].Print();
			cout << " ";

			int nextLayer = x + 1;
			if(nextLayer < LAYERS)
			{
				int block = layers[nextLayer]->Length;
				for(int w = 0; w < block; w++)
				{
					int index = y * block + w;

					cout << y << "->" << w;

					weights[x]->HostData[index].Print();

					cout << " ";
				}
			}
		}

		cout << "\n";
	}

//	for(int i = 0; i < layer0->Length && i < 512; i++)
//	{
//		cout << "Node.Self " << layer0->HostData[i].Self.Value << " Derivative " << layer0->HostData[i].Self.Derivative << "\n";
//		cout << "Node.Bias " << layer0->HostData[i].Bias.Value << " Derivative " << layer0->HostData[i].Bias.Derivative << "\n";
//
//		for(int x = 0; x < layer1->Length; x++)
//		{
//			int w = i * layer1->Length + x;
//
//			cout << "Weight.Self " << weights->HostData[w].Value << " Derivative " << weights->HostData[w].Derivative << "\n";
//		}
//	}

    cout << "print finished\n";

//	leftValues->Dispose();
//	weights->Dispose();
//	rightValues->Dispose();
//	rightBiases->Dispose();

	for(int i = 0; i < layers.size(); i++)
	{
		delete layers[i];
	}

	for(int i = 0; i < weights.size(); i++)
	{
		delete weights[i];
	}

//	delete layers[1];
//	delete weight0to1;
//	delete layer1;

    cout << "dispose finished\n";

    hipDeviceReset();

    cout << "hipDeviceReset finished\n";

    cout << "will exit\n";

//    exit(EXIT_SUCCESS);

	return 0;
}

inline double randomValue()
{
	return 0.5 + (double)rand() / RAND_MAX;
}

void randomValues(double* a, int n)
{
	int i;
	for (i = 0; i < n; ++i)
	{
		a[i] = randomValue();

//		cout << "random = " << a[i] << "\n";
	}
}

void randomValues(Node* a, int n)
{
	int i;
	for (i = 0; i < n; ++i)
	{
		a[i].Self.Value = randomValue();
		a[i].Bias.Value = randomValue();
	}
}

void randomValues(Element* a, int n)
{
	int i;
	for (i = 0; i < n; ++i)
	{
		a[i].Value = randomValue();
	}
}

