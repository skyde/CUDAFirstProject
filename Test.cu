#include "hip/hip_runtime.h"

//#include "GL/glut.h"
// OpenGL Graphics includes
//#include <GL/glew.h>
//#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
//#include <GL/wglew.h>
//#endif
//#if defined(__APPLE__) || defined(__MACOSX)
//  #pragma clang diagnostic ignored "-Wdeprecated-declarations"
//  #include <GLUT/glut.h>
//  #ifndef glutCloseFunc
//  #define glutCloseFunc glutWMCloseFunc
//  #endif
//#else
//#include <GL/freeglut.h>
//#endif
//
//// CUDA runtime
//// CUDA utilities and system includes
//#include <hip/hip_runtime.h>
//#include <cuda_gl_interop.h>
//
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>
//#include <helper_cuda_gl.h>
//#include <rendercheck_gl.h>


#include <stdio.h>
#include <iostream>

#include "SharedData.cu"
using namespace std;

void random_ints(int* a, int n);
//void initGL(int *argc, char **argv);

// Total Threads
#define N (4096 * 8)
// Block Size
#define M 512

#define RADIUS 1

class Layer
{
public:

};

//__global__ void add(int *a, int *b, int *c, int n)
//{
//	int index = threadIdx.x + blockIdx.x * blockDim.x;
//
//	if(index < n)
//	{
//		c[index] = a[index] + b[index];
//	}
//}

__global__ void stencil_1d(int* in, int* out)
{
	__shared__ int temp[M + 2 * RADIUS];
	int gindex = threadIdx.x + blockIdx.x * blockDim.x;
	int lindex = threadIdx.x + RADIUS;

	temp[lindex] = in[gindex];

	if(threadIdx.x < RADIUS)
	{
		temp[lindex - RADIUS] = in[gindex - RADIUS];
		temp[lindex + M] = in[gindex + M];
	}

	__syncthreads();

	int result = 0;

	for	(int offset = -RADIUS; offset <= RADIUS; offset++)
	{
		result += temp[lindex + offset];
	}

	out[gindex] = result;
}

//template <class T>
//class Stack {
//};

int main(int argc, char **argv)
{
//	initGL(&argc, argv);
	printf ("N = %d \n", N);

	SharedData<int>* a = new SharedData<int>(N);
	SharedData<int>* b = new SharedData<int>(N);
//	SharedData<int>* c = new SharedData<int>(N);

//	double *inputs;
//	double *outputs;

//	int *a, *b, *c;
//	int *d_a, *d_b, *d_c;
//	int size = N * sizeof(int);

//	hipMalloc((void **)&d_a, size);
//	hipMalloc((void **)&d_b, size);
//	hipMalloc((void **)&d_c, size);
//
//	a = (int *)malloc(size);
//	b = (int *)malloc(size);
//	c = (int *)malloc(size);

	random_ints(a->HostData, a->Length);
//	random_ints(b->HostData, N);

	a->CopyToDevice();
//	b->CopyToDevice();

//	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
//	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	stencil_1d<<<((N + M - 1) / M), M>>>(a->DeviceData, b->DeviceData);

	b->CopyToHost();
//	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	for(int i = 0; i < b->Length && i < 512; i++)
	{
		printf ("%d = %d \n", a->HostData[i], b->HostData[i]);
	}

//	free(a);
//	free(b);
//	free(c);

//	hipFree(d_a);
//	hipFree(d_b);
//	hipFree(d_c);

	delete a;
	delete b;

    hipDeviceReset();
    exit(EXIT_SUCCESS);

	return 0;
}
//
//void initGL(int *argc, char **argv)
//{
////    printf("Initializing GLUT...\n");
////    glutInit(argc, argv);
////
////    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
////    glutInitWindowSize(1024, 768);
////    glutInitWindowPosition(0, 0);
////    glutCreateWindow(argv[0]);
////
//////    glutDisplayFunc(displayFunc);
//////    glutKeyboardFunc(keyboardFunc);
//////    glutMouseFunc(clickFunc);
//////    glutMotionFunc(motionFunc);
//////    glutReshapeFunc(reshapeFunc);
//////    glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
//////    initMenus();
////
////    printf("Loading extensions: %s\n", glewGetErrorString(glewInit()));
////
////    if (!glewIsSupported("GL_VERSION_1_5 GL_ARB_vertex_buffer_object GL_ARB_pixel_buffer_object"))
////    {
////        fprintf(stderr, "Error: failed to get minimal extensions for demo\n");
////        fprintf(stderr, "This sample requires:\n");
////        fprintf(stderr, "  OpenGL version 1.5\n");
////        fprintf(stderr, "  GL_ARB_vertex_buffer_object\n");
////        fprintf(stderr, "  GL_ARB_pixel_buffer_object\n");
////        exit(EXIT_SUCCESS);
////    }
////
////    printf("OpenGL window created.\n");
//}
void random_ints(int* a, int n)
{
	int i;
	for (i = 0; i < n; ++i)
	{
		a[i] = rand() % 4;
	}
}
