#include "hip/hip_runtime.h"

//#include "GL/glut.h"
// OpenGL Graphics includes
//#include <GL/glew.h>
//#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
//#include <GL/wglew.h>
//#endif
//#if defined(__APPLE__) || defined(__MACOSX)
//  #pragma clang diagnostic ignored "-Wdeprecated-declarations"
//  #include <GLUT/glut.h>
//  #ifndef glutCloseFunc
//  #define glutCloseFunc glutWMCloseFunc
//  #endif
//#else
//#include <GL/freeglut.h>
//#endif
//
//// CUDA runtime
//// CUDA utilities and system includes
//#include <hip/hip_runtime.h>
//#include <cuda_gl_interop.h>
//
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>
//#include <helper_cuda_gl.h>
//#include <rendercheck_gl.h>


#include <stdio.h>
#include <iostream>
#include "hip/hip_runtime_api.h"

#include "SharedData.cu"
using namespace std;

void randomValues(double* a, int n);
//void initGL(int *argc, char **argv);

// Total Threads
#define N 16 // 4096
// Block Size
#define M 2 // 512

#define RADIUS 1

class Layer
{
public:

};

//__global__ void add(int *a, int *b, int *c, int n)
//{
//	int index = threadIdx.x + blockIdx.x * blockDim.x;
//
//	if(index < n)
//	{
//		c[index] = a[index] + b[index];
//	}
//}

__global__ void ForwardPass(
		double* source,
		double* weights,
		double* target,
		double* biases)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	double output = source[index];

	for(int i = 0; i < N; i++)
	{
		output *= weights[index * N + i];
	}

	target[index] = tanh(output + biases[index]);
}

//__global__ void BackwardPass(double* source, double* weights, double* target)
//{
//	int index = threadIdx.x + blockIdx.x * blockDim.x;
//
//	double output = source[index];
//
//	for(int i = 0; i < N; i++)
//	{
//		output *= weights[index * N + i];
//	}
//
//	target[index] = tanh(output);
//}

int main(int argc, char **argv)
{
	printf ("N = %d \n", N);

	SharedData* sourceLayer = new SharedData(N);
	SharedData* weights = new SharedData(N * N);
	SharedData* nextLayer = new SharedData(N);
	SharedData* nextLayerBiases = new SharedData(N);

	randomValues(sourceLayer->HostData, sourceLayer->Length);
	randomValues(weights->HostData, weights->Length);
	randomValues(nextLayerBiases->HostData, nextLayerBiases->Length);

	cout << "Generated random values\n";

	sourceLayer->CopyToDevice();
	weights->CopyToDevice();
	nextLayerBiases->CopyToDevice();

	cout << "Copy to device calls after initiated\n";

//	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
//	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

//	dim3 threadsPerBlock(16, 16);
//	dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);

	ForwardPass<<<N / M, M>>>(
			sourceLayer->DeviceData,
			weights->DeviceData,
			nextLayer->DeviceData,
			nextLayerBiases->DeviceData);

	cout << "RunPass initiated\n";

	nextLayer->CopyToHost();

	cout << "CopyToHost initiated\n";

    hipDeviceSynchronize();

	cout << "hipDeviceSynchronize finished\n";

    getLastCudaError("Device kernel execution failed.\n");

    cout << "Execution finished, will print\n";

	for(int i = 0; i < nextLayer->Length && i < 512; i++)
	{
		cout << sourceLayer->HostData[i] << " = " << nextLayer->HostData[i] << "\n";
	}

    cout << "print finished\n";

	sourceLayer->Dispose();
	weights->Dispose();
	nextLayer->Dispose();
	nextLayerBiases->Dispose();

    cout << "dispose finished\n";

    hipDeviceReset();

    cout << "hipDeviceReset finished\n";

	delete sourceLayer;
	delete weights;
	delete nextLayer;
	delete nextLayerBiases;

    cout << "will exit\n";

//    exit(EXIT_SUCCESS);

	return 0;
}

void randomValues(double* a, int n)
{
	int i;
	for (i = 0; i < n; ++i)
	{
		a[i] = 0.5 + (double)rand() / RAND_MAX;

		cout << "random = " << a[i] << "\n";
	}
}
