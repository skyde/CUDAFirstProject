#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include "hip/hip_runtime_api.h"

#include "SharedData.cu"
#include "Layer.cu"
using namespace std;

//void initGL(int *argc, char **argv);

// Total Threads
#define N 2 // 4096
// Block Size
#define M 2 // 512

#define RADIUS 1


//__global__ void add(int *a, int *b, int *c, int n)
//{
//	int index = threadIdx.x + blockIdx.x * blockDim.x;
//
//	if(index < n)
//	{
//		c[index] = a[index] + b[index];
//	}
//}

struct __align__(sizeof(double) * 2) Element
{
    double Value, Derivative;
};

struct __align__(sizeof(Element) * 2) Node
{
	Element Self, Bias;
};

void randomValues(double* a, int n);
void randomValues(Node* a, int n);
void randomValues(Element* a, int n);

__global__ void ForwardPass(
		Node* left,
		Element* weights, // left to right
		Node* right)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	double output = left[index].Self.Value;

	for(int i = 0; i < N; i++)
	{
		output *= weights[index * N + i].Value;
	}

	right[index].Self.Value = tanh(output + right[index].Bias.Value);
}

int main(int argc, char **argv)
{
	printf ("N = %d \n", N);

	SharedData<Node>* layer0 = new SharedData<Node>(N);
	SharedData<Element>* weights = new SharedData<Element>(N * N);
	SharedData<Node>* layer1 = new SharedData<Node>(N);

	randomValues(layer0->HostData, layer0->Length);
	randomValues(weights->HostData, weights->Length);
	randomValues(layer1->HostData, layer1->Length);

	cout << "Generated random values\n";

	layer0->CopyToDevice();
	weights->CopyToDevice();
	layer1->CopyToDevice();

	cout << "Copy to device calls after initiated\n";

	ForwardPass<<<N / M, M>>>(
			layer0->DeviceData,
			weights->DeviceData,
			layer1->DeviceData);

	cout << "RunPass initiated\n";

	layer1->CopyToHost();

	cout << "CopyToHost initiated\n";

    hipDeviceSynchronize();

	cout << "hipDeviceSynchronize finished\n";

    getLastCudaError("Device kernel execution failed.\n");

    cout << "Execution finished, will print\n";

	for(int i = 0; i < layer0->Length && i < 512; i++)
	{
		cout << "Node.Self " << layer0->HostData[i].Self.Value << " Derivative " << layer0->HostData[i].Self.Derivative << "\n";
		cout << "Node.Bias " << layer0->HostData[i].Bias.Value << " Derivative " << layer0->HostData[i].Bias.Derivative << "\n";

		for(int x = 0; x < layer1->Length; x++)
		{
			int w = i * layer1->Length + x;

			cout << "Weight.Self " << weights->HostData[w].Value << " Derivative " << weights->HostData[w].Derivative << "\n";
		}
	}

    cout << "print finished\n";

//	leftValues->Dispose();
//	weights->Dispose();
//	rightValues->Dispose();
//	rightBiases->Dispose();

	delete layer0;
	delete weights;
	delete layer1;

    cout << "dispose finished\n";

    hipDeviceReset();

    cout << "hipDeviceReset finished\n";

    cout << "will exit\n";

//    exit(EXIT_SUCCESS);

	return 0;
}

inline double randomValue()
{
	return 0.5 + (double)rand() / RAND_MAX;
}

void randomValues(double* a, int n)
{
	int i;
	for (i = 0; i < n; ++i)
	{
		a[i] = randomValue();

//		cout << "random = " << a[i] << "\n";
	}
}

void randomValues(Node* a, int n)
{
	int i;
	for (i = 0; i < n; ++i)
	{
		a[i].Self.Value = randomValue();
		a[i].Bias.Value = randomValue();
	}
}

void randomValues(Element* a, int n)
{
	int i;
	for (i = 0; i < n; ++i)
	{
		a[i].Value = randomValue();
	}
}

