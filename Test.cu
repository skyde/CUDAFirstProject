
#include <hip/hip_runtime.h>
#include <stdio.h>
//using namespace std;

void random_ints(int* a, int n);

#define N (2048 * 2048)
#define M 512

__global__ void add(int *a, int *b, int *c, int n)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	if(index < n)
	{
		c[index] = a[index] + b[index];
	}
}

int main(void)
{
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof(int);

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);

	random_ints(a, N);
	random_ints(b, N);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	add<<<((N + M - 1) / M), M>>>(d_a, d_b, d_c, N);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	for(int i = 0; i < N && i < 512; i++)
	{
		printf ("%d + %d = %d \n", *(a + i), *(b + i), *(c + i));
	}

	free(a);
	free(b);
	free(c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

//	cout << "test";
	return 0;
}

void random_ints(int* a, int n)
{
	int i;
	for (i = 0; i < n; ++i)
	{
		a[i] = rand() % 50;
	}
}
