#include "hip/hip_runtime.h"

//#include "GL/glut.h"
// OpenGL Graphics includes
//#include <GL/glew.h>
//#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
//#include <GL/wglew.h>
//#endif
//#if defined(__APPLE__) || defined(__MACOSX)
//  #pragma clang diagnostic ignored "-Wdeprecated-declarations"
//  #include <GLUT/glut.h>
//  #ifndef glutCloseFunc
//  #define glutCloseFunc glutWMCloseFunc
//  #endif
//#else
//#include <GL/freeglut.h>
//#endif
//
//// CUDA runtime
//// CUDA utilities and system includes
//#include <hip/hip_runtime.h>
//#include <cuda_gl_interop.h>
//
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>
//#include <helper_cuda_gl.h>
//#include <rendercheck_gl.h>


#include <stdio.h>
#include <iostream>
#include "hip/hip_runtime_api.h"

#include "SharedData.cu"
using namespace std;

void randomValues(double* a, int n);
//void initGL(int *argc, char **argv);

// Total Threads
#define N 16 // 4096
// Block Size
#define M 2 // 512

#define RADIUS 1

class Layer
{
public:

};

//__global__ void add(int *a, int *b, int *c, int n)
//{
//	int index = threadIdx.x + blockIdx.x * blockDim.x;
//
//	if(index < n)
//	{
//		c[index] = a[index] + b[index];
//	}
//}

__global__ void RunPass(double* source, double* weights, double* target)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	double output = source[index];

	for(int i = 0; i < N; i++)
	{
		output *= weights[index * N + i];
	}

	target[index] = output * 2;
//	__shared__ int temp[M + 2 * RADIUS];
//	int gindex = threadIdx.x + blockIdx.x * blockDim.x;
//	int lindex = threadIdx.x + RADIUS;
//
//	temp[lindex] = in[gindex];
//
//	if(threadIdx.x < RADIUS)
//	{
//		temp[lindex - RADIUS] = in[gindex - RADIUS];
//		temp[lindex + M] = in[gindex + M];
//	}
//
//	__syncthreads();
//
//	int result = 0;
//
//	for	(int offset = -RADIUS; offset <= RADIUS; offset++)
//	{
//		result += temp[lindex + offset];
//	}
//
//	out[gindex] = result;
}

//template <class T>
//class Stack {
//};

int main(int argc, char **argv)
{
//	initGL(&argc, argv);
	printf ("N = %d \n", N);

	SharedData* layer0 = new SharedData(N);
	SharedData* weights = new SharedData(N * N);
	SharedData* layer1 = new SharedData(N);

//    hipDeviceSynchronize();

	randomValues(layer0->HostData, layer0->Length);
	randomValues(weights->HostData, weights->Length);
//	random_ints(b->HostData, N);

	cout << "Generated random values\n";

	layer0->CopyToDevice();
	weights->CopyToDevice();
//	b->CopyToDevice();
	cout << "Copy to device calls after initiated\n";

//	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
//	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

//	dim3 threadsPerBlock(16, 16);
//	dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);

	RunPass<<<N / M, M>>>(layer0->DeviceData, weights->DeviceData, layer1->DeviceData);

	cout << "RunPass initiated\n";
//    hipError_t err = hipGetLastError();
//
//    if (hipSuccess != err)
//    {
//        fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
//                file, line, "Execution on device failed", (int)err, hipGetErrorString(err));
//        DEVICE_RESET
//        exit(EXIT_FAILURE);
//    }

//    hipDeviceSynchronize();

	layer1->CopyToHost();

	cout << "CopyToHost initiated\n";
//	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
	cout << "hipDeviceSynchronize finished\n";

    getLastCudaError("Device kernel execution failed.\n");

    cout << "Execution finished, will print\n";

	for(int i = 0; i < layer1->Length && i < 512; i++)
	{
		cout << layer0->HostData[i] << " = " << layer1->HostData[i] << "\n";
	}

    cout << "print finished\n";

	layer0->Dispose();
	weights->Dispose();
	layer1->Dispose();

    cout << "dispose finished\n";

//    hipDeviceReset();
    cout << "hipDeviceReset finished\n";
    exit(EXIT_SUCCESS);

	delete layer0;
	delete weights;
	delete layer1;

    cout << "will exit\n";

	return 0;
}

void randomValues(double* a, int n)
{
	int i;
	for (i = 0; i < n; ++i)
	{
		a[i] = 1.0 + (double)rand() / RAND_MAX;

		cout << "random = " << a[i] << "\n";
	}
}
