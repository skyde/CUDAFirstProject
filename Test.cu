#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include "hip/hip_runtime_api.h"

#include "SharedData.cu"
#include "Layer.cu"
using namespace std;

//void initGL(int *argc, char **argv);

// Total Threads
#define N 16 // 4096
// Block Size
#define M 2 // 512

#define RADIUS 1


//__global__ void add(int *a, int *b, int *c, int n)
//{
//	int index = threadIdx.x + blockIdx.x * blockDim.x;
//
//	if(index < n)
//	{
//		c[index] = a[index] + b[index];
//	}
//}

struct __align__(sizeof(double) * 2) Element
{
    double Value, Derivative;
};

struct __align__(sizeof(Element) * 2) Node
{
	Element Self, Bias;
};

void randomValues(double* a, int n);
void randomValues(Node* a, int n);
void randomValues(Element* a, int n);

__global__ void ForwardPass(
		Node* left,
		Element* weights, // left to right
		Node* right)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	double output = left[index].Self.Value;

	for(int i = 0; i < N; i++)
	{
		output *= weights[index * N + i].Value;
	}

	right[index].Self.Value = tanh(output + right[index].Bias.Value);
}

//__global__ void BackwardPass(
//		double* leftValues,
//		double* leftValuesDerivatives,
//		double* leftBiases,
//		double* leftBiasesDerivatives,
//		double* weights,
//		double* weightsDerivatives,
//		double* rightValues,
//		double* rightValuesDerivatives,
//		double* rightBiasesDerivatives)
//{
////	int index = threadIdx.x + blockIdx.x * blockDim.x;
////
////	double output = leftValues[index];
////
////	for(int i = 0; i < N; i++)
////	{
////		output *= weights[index * N + i];
////	}
////
////	rightValues[index] = tanh(output + rightBiases[index]);
//}
//__global__ void BackwardPass(
//		double* left,
//		double* weights, // left to right
//		double* right,
//		double* biases)
//{
//	int index = threadIdx.x + blockIdx.x * blockDim.x;
//
//	double output = left[index];
//
//	for(int i = 0; i < N; i++)
//	{
//		output *= weights[index * N + i];
//	}
//
//	right[index] = tanh(output + biases[index]);
//}

int main(int argc, char **argv)
{
	printf ("N = %d \n", N);

	SharedData<Node>* layer0 = new SharedData<Node>(N);
	SharedData<Element>* weights = new SharedData<Element>(N * N);
	SharedData<Node>* layer1 = new SharedData<Node>(N);

//	SharedData<double>* leftValues = new SharedData<double>(N);
//	SharedData<double>* weights = new SharedData<double>(N * N);
//	SharedData<double>* rightValues = new SharedData<double>(N);
//	SharedData<double>* rightBiases = new SharedData<double>(N);

	randomValues(layer0->HostData, layer0->Length);
	randomValues(weights->HostData, weights->Length);
	randomValues(layer1->HostData, layer1->Length);

	cout << "Generated random values\n";

	layer0->CopyToDevice();
	weights->CopyToDevice();
	layer1->CopyToDevice();

	cout << "Copy to device calls after initiated\n";

//	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
//	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

//	dim3 threadsPerBlock(16, 16);
//	dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);

	ForwardPass<<<N / M, M>>>(
			layer0->DeviceData,
			weights->DeviceData,
			layer1->DeviceData);

	cout << "RunPass initiated\n";

	layer1->CopyToHost();

	cout << "CopyToHost initiated\n";

    hipDeviceSynchronize();

	cout << "hipDeviceSynchronize finished\n";

    getLastCudaError("Device kernel execution failed.\n");

    cout << "Execution finished, will print\n";

	for(int i = 0; i < layer0->Length && i < 512; i++)
	{
		cout << layer0->HostData[i].Self.Value << " = " << layer1->HostData[i].Self.Value << "\n";
	}

    cout << "print finished\n";

//	leftValues->Dispose();
//	weights->Dispose();
//	rightValues->Dispose();
//	rightBiases->Dispose();

	delete layer0;
	delete weights;
	delete layer1;

    cout << "dispose finished\n";

    hipDeviceReset();

    cout << "hipDeviceReset finished\n";

    cout << "will exit\n";

//    exit(EXIT_SUCCESS);

	return 0;
}

inline double randomValue()
{
	return 0.5 + (double)rand() / RAND_MAX;
}

void randomValues(double* a, int n)
{
	int i;
	for (i = 0; i < n; ++i)
	{
		a[i] = randomValue();

//		cout << "random = " << a[i] << "\n";
	}
}

void randomValues(Node* a, int n)
{
	int i;
	for (i = 0; i < n; ++i)
	{
		a[i].Self.Value = randomValue();
		a[i].Bias.Value = randomValue();
	}
}

void randomValues(Element* a, int n)
{
	int i;
	for (i = 0; i < n; ++i)
	{
		a[i].Value = randomValue();
	}
}

