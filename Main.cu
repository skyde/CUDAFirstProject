#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <array>
using namespace std;

#include "NeuralNetwork.cu"
__global__ void ForwardPass(
		Node* left,
		int leftLength,
		Element* weights,
		Node* right,
		int rightLength)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	double value = 0;

	for(int i = 0; i < leftLength; i++)
	{
		int w = index + i * rightLength;

		value += left[i].Self.Value * weights[w].Value;
	}

	right[index].Self.Value = value + right[index].Bias.Value;
}

__global__ void BackwardPass(
		Node* left,
		int leftLength,
		Element* weights,
		Node* right,
		int rightLength)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	double total = 0;

	double leftValue = left[index].Self.Value;

	for(int i = 0; i < rightLength; i++)
	{
		int w = index * rightLength + i;
		double rightDerivative = right[i].Self.Derivative;

		weights[w].Derivative = leftValue * rightDerivative;//weights[w].Value * right[i].Self.Derivative;

		total += weights[w].Value * rightDerivative;
	}

	left[index].Self.Derivative = total;
}

__global__ void CaculateDerivativesFromDifferencePass(
		Node* values,
		double* targets)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	double v = targets[index] - values[index].Self.Value;

	values[index].Self.Derivative = v * 0.01;
}

__global__ void IterateDerivativePass(Element* weights)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	weights[index].Value += weights[index].Derivative;
//	left[index].Self.Derivative = total;
}

void Forward(NeuralNetwork* n)
{
	for(int i = 0; i < n->Layers.size() - 1; ++i)
	{
		ForwardPass<<<N / M, M>>>(
				n->Layers[i]->DeviceData,
				n->Layers[i]->Length,
				n->Weights[i]->DeviceData,
				n->Layers[i + 1]->DeviceData,
				n->Layers[i + 1]->Length);
	}
}

void CaculateDerivativesFromDifference(NeuralNetwork* n, SharedData<double>* targetValues)
{
	SharedData<Node>* layer = n->Layers[n->Layers.size() - 1];
	int length = layer->Length;

	CaculateDerivativesFromDifferencePass<<<length, 1>>>(
			layer->DeviceData,
			targetValues->DeviceData);
}

void Backward(NeuralNetwork* n)
{
	BackwardPass<<<N / M, M>>>(
			n->Layers[0]->DeviceData,
			n->Layers[0]->Length,
			n->Weights[0]->DeviceData,
			n->Layers[1]->DeviceData,
			n->Layers[1]->Length);
}

void IterateDerivative(NeuralNetwork* n)
{
	int length = n->Weights[0]->Length;
	IterateDerivativePass<<<length / M, M>>>(
			n->Weights[0]->DeviceData);
}

int main(int argc, char **argv)
{
	NeuralNetwork* n = new NeuralNetwork();

	n->Layers[0]->HostData[0].Self.Value = 1;
	n->Layers[0]->HostData[1].Self.Value = 2;

	n->Layers[1]->HostData[0].Self.Derivative = 0.01;
	n->Layers[1]->HostData[1].Self.Derivative = 0.001;

	n->Weights[0]->HostData[0].Value = 1;
	n->Weights[0]->HostData[1].Value = 0.5;
	n->Weights[0]->HostData[2].Value = 0;
	n->Weights[0]->HostData[3].Value = 1;

	n->CopyToDevice();

//	double targetValues[2] = { 4.0, -2.0 };

	SharedData<double>* targetValues = new SharedData<double>(N);
	targetValues->HostData[0] = 4;
	targetValues->HostData[1] = -2;
	targetValues->CopyToDevice();

//	cout << "Copy to device calls after initiated\n";

	for(int i = 0; i < 1000; ++i)
	{
		cout << "\n";
		cout << "Epoch " << i;
		cout << "\n";

		Forward(n);
		CaculateDerivativesFromDifference(n, targetValues);
		Backward(n);
		IterateDerivative(n);

		n->CopyToHost();

	    hipDeviceSynchronize();

	    n->Print();
	}

//	cout << "RunPass initiated\n";


//	cout << "CopyToHost initiated\n";

//    hipDeviceSynchronize();

//	cout << "hipDeviceSynchronize finished\n";

//    getLastCudaError("Device kernel execution failed.\n");

//    cout << "Execution finished, will print\n";

//    cout << "print finished\n";

    delete n;
    delete targetValues;

    cout << "dispose finished\n";

    hipDeviceReset();

    cout << "hipDeviceReset finished\n";

    cout << "will exit\n";

//    exit(EXIT_SUCCESS);

	return 0;
}


