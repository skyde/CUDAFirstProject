#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <array>
#include "NeuralNetwork.cu"
using namespace std;
#include <vector>
#include <fstream>
#include "ReadMNIST.cpp"
#include "MNISTData.cu"

//int main()
//{
//  vector<vector<double>> ar;
//  ReadMNIST(10000,784,ar);
//
//  return 0;
//}

__global__ void ForwardPass(
		Node* left,
		int leftLength,
		Element* weights,
		Node* right,
		int rightLength)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	double value = 0;

	for(int i = 0; i < leftLength; i++)
	{
		int w = index + i * rightLength;

		value += left[i].Self.Value * weights[w].Value;
	}

	value += right[index].Bias.Value;

	if(right[index].Activation == ActivationTanH)
	{
		value = tanh(value);
	}

	right[index].Self.Value = value;
}

__global__ void BackwardPass(
		Node* left,
		int leftLength,
		Element* weights,
		Node* right,
		int rightLength)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	double value = 0;

	double leftValue = left[index].Self.Value;

	for(int i = 0; i < rightLength; i++)
	{
		int w = index * rightLength + i;
		double rightDerivative = right[i].Self.Derivative;

		weights[w].Derivative = leftValue * rightDerivative;//weights[w].Value * right[i].Self.Derivative;

		value += weights[w].Value * rightDerivative;
	}

	left[index].Bias.Derivative = value;

	if(left[index].Activation == ActivationTanH)
	{
		value = 2.0 / (1.0 + pow(2.718281828459, -2.0 * value)) - 1.0;//tanh(value);
	}

	left[index].Self.Derivative = value;
}

__global__ void CaculateDerivativesFromDifferencePass(
		Node* values,
		double* targets)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	double v = targets[index] - values[index].Self.Value;

	values[index].Self.Derivative = v * STEP_SIZE;
}

__global__ void IterateWeightDerivativePass(Element* weights)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	weights[index].Value += weights[index].Derivative;
}

__global__ void IterateNodeDerivativePass(Node* nodes)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

//	nodes[index].Self.Value += nodes[index].Self.Derivative;
	nodes[index].Bias.Value += nodes[index].Bias.Derivative;
}

__global__ void SetInputValuesPass(
		Node* values,
		double* targets)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	values[index].Self.Value = targets[index];
}

void Forward(NeuralNetwork* n)
{
	for(int i = 0; i < n->Layers.size() - 1; ++i)
	{
		int leftLength = n->Layers[i]->Length;
		int rightLength = n->Layers[i + 1]->Length;

//		cout << "\n" << leftLength << "\n";

		ForwardPass<<<rightLength / M, M>>>(
				n->Layers[i]->DeviceData,
				leftLength,
				n->Weights[i]->DeviceData,
				n->Layers[i + 1]->DeviceData,
				rightLength);
	}
}

void CaculateDerivativesFromDifference(NeuralNetwork* n, SharedData<double>* targetValues)
{
	SharedData<Node>* layer = n->Layers[n->Layers.size() - 1];
	int length = layer->Length;

	CaculateDerivativesFromDifferencePass<<<length, 1>>>(
			layer->DeviceData,
			targetValues->DeviceData);
}

void SetInputValues(NeuralNetwork* n, SharedData<double>* values)
{
	SharedData<Node>* layer = n->Layers[0];
	int length = layer->Length;

	SetInputValuesPass<<<length, 1>>>(
			layer->DeviceData,
			values->DeviceData);
}

void Backward(NeuralNetwork* n)
{
	for(int i = n->Layers.size() - 2; i >= 0; --i)
	{
		int leftLength = n->Layers[i]->Length;
		int rightLength = n->Layers[i + 1]->Length;

		BackwardPass<<<leftLength / M, M>>>(
				n->Layers[i]->DeviceData,
				leftLength,
				n->Weights[i]->DeviceData,
				n->Layers[i + 1]->DeviceData,
				rightLength);
	}
}

void IterateDerivative(NeuralNetwork* n)
{
	for(int i = 0; i < n->Weights.size(); ++i)
	{
		int length = n->Weights[i]->Length;
		IterateWeightDerivativePass<<<length, 1>>>(
				n->Weights[i]->DeviceData);
	}

	// Don't iterate input or output layer
	for(int i = 1; i < n->Layers.size() - 1; ++i)
	{
		int length = n->Layers[i]->Length;
		IterateNodeDerivativePass<<<length, 1>>>(
				n->Layers[i]->DeviceData);
	}
}

void SetData(array<MNISTData*, 10> data, NeuralNetwork* n, int number, int currentElement)
{
	SetInputValues(n, data[number]->Elements[currentElement % MNIST_ELEMENTS_TO_LOAD]->Data);
	SharedData<double>* targetValues = data[number]->TargetValues;
}

void CaculateAccuracy(array<MNISTData*, 10> data, NeuralNetwork* n)
{
	double total = 0.0;
	double totalNum = 0.0;

	for(int i = 0; i < 10; ++i)
	{
		for(int x = 0; x < data.size(); ++x)
		{
			SetData(data, n, x, i);

			Forward(n);

			n->Layers[LAYERS - 1]->CopyToHost();

			hipDeviceSynchronize();

			int index = -1;
			double largest = -1;

			for(int v = 0; v < n->Layers[LAYERS - 1]->Length; v++)
			{
				double value = n->Layers[LAYERS - 1]->HostData[v].Self.Value;

				if(value > largest)
				{
					largest = value;
					index = v;
				}
			}

			bool correct = index == x ? true : false;

			if(correct)
			{
				total++;
			}

			totalNum++;

		}
	}

	double accuracy = total / totalNum;

	cout << "Accuracy " << accuracy << "\n";
}

int main(int argc, char **argv)
{
	cout << "Reading in MNIST\n";

//	array<double*, MNIST_ELEMENTS_TO_LOAD> values = ReadMNISTData("data0.txt");

	array<MNISTData*, 10> data;

	for(int i = 0; i < data.size(); ++i)
	{
		data[i] = new MNISTData(i);
	}
//	vector< vector<double> > ar;
//	ReadMNIST(10000, 784, ar);

	cout << "End reading in MNIST\n";

//	return 0;

	NeuralNetwork* n = new NeuralNetwork();

//	n->Layers[0]->HostData[0].Self.Value = 1;
//	n->Layers[0]->HostData[1].Self.Value = 5;
//	n->Layers[0]->HostData[2].Self.Value = -5;

	// Middle layers are activated
	for(int i = 1; i < LAYERS - 1; ++i)
	{
		n->Layers[1]->HostData[i].Activation = ActivationTanH;
	}

	n->CopyToDevice();

	cout << "\n";

	int currentElement = 0;

	for(int i = 0; i < 1000000; ++i)
	{
		cout << "Epoch " << (i + 1);

		for(int x = 0; x < data.size(); ++x)
		{
			SetData(data, n, x, currentElement);

			Forward(n);
			CaculateDerivativesFromDifference(n, data[x]->TargetValues);
			Backward(n);
			IterateDerivative(n);

//			if(PRINT_ERROR)
//			{
//				n->CopyToHost();

//				cout << ", error " << n->CaculateError(targetValues->HostData, false);
//
//				cout << " ";
//
//				n->CaculateError(targetValues->HostData, true);
//			}

			if(PRINT_VERBOSE)
			{
				n->PrintVerbose();
			}
		}
		cout << "\n";


		hipDeviceSynchronize();

		if(i % 10 == 0)
		{
			CaculateAccuracy(data, n);
		}

		currentElement++;
	}


//	#if !PRINT_VERBOSE
//    n->PrintVerbose();
//	#endif

//	cout << "RunPass initiated\n";


//	cout << "CopyToHost initiated\n";

//    hipDeviceSynchronize();

//	cout << "hipDeviceSynchronize finished\n";

//    getLastCudaError("Device kernel execution failed.\n");

//    cout << "Execution finished, will print\n";

//    cout << "print finished\n";

    delete n;
	for(int i = 0; i < data.size(); ++i)
	{
		delete data[i];
	}

    cout << "dispose finished\n";

    hipDeviceReset();

    cout << "hipDeviceReset finished\n";

    cout << "will exit\n";

//    exit(EXIT_SUCCESS);

	return 0;
}


