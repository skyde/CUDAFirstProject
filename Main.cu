#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <array>
#include "hip/hip_runtime_api.h"
#include <stdlib.h>

#include "SharedData.cu"
#include "Layer.cu"
#include "Element.cu"
#include "Node.cu"
using namespace std;

// Total Threads
#define N 2 // Nodes per layer
// Block Size
#define M 1 // 512

#define LAYERS 2

#define PRINT_DERIVATIVE true

void randomValues(double* a, int n);
void randomValues(Node* a, int n);
void randomValues(Element* a, int n);

__global__ void ForwardPass(
		Node* left,
		int leftLength,
		Element* weights, // left to right
		Node* right,
		int rightLength)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	double value = 0;//left[index].Self.Value;

	for(int i = 0; i < leftLength; i++)
	{
		int w = index + i * rightLength;

		value += left[i].Self.Value * weights[w].Value;

//		weights[w].Derivative = -9;
	}

	right[index].Self.Value = value;
//	right[index].Self.Derivative = -7;// + right[index].Bias.Value
}

__global__ void BackwardPass(
		Node* left,
		int leftLength,
		Element* weights,
		Node* right,
		int rightLength)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

//	double rightDerivative = right[index].Self.Derivative;

	double total = 0;

	for(int i = 0; i < rightLength; i++)
	{
		int w = index * rightLength + i;

		weights[w].Derivative = weights[w].Value * right[i].Self.Derivative;

		total += weights[w].Derivative;
//		value += left[i].Self.Value * weights[w].Value;
	}

	left[index].Self.Derivative = total;

//	__syncthreads();

//	int index = threadIdx.x + blockIdx.x * blockDim.x;
//
//	double output = left[index].Self.Value;
//
//	for(int i = 0; i < N; i++)
//	{
//		output *= weights[index * N + i].Value;
//	}
//
//	right[index].Self.Value = tanh(output + right[index].Bias.Value);
}

int main(int argc, char **argv)
{
	printf ("N = %d \n", N);

	array<SharedData<Node>*, LAYERS> layers;
	array<SharedData<Element>*, LAYERS - 1> weights;

	for(int i = 0; i < layers.size(); i++)
	{
		layers[i] = new SharedData<Node>(N);

		if(i != 0)
		{
			int length = layers[i - 1]->Length * layers[i]->Length;

			weights[i - 1] = new SharedData<Element>(length);

			cout << "weights " << i - 1 << ", l = " << length << "\n";
		}
	}

	layers[0]->HostData[0].Self.Value = 1;
	layers[0]->HostData[1].Self.Value = 2;

	layers[1]->HostData[0].Self.Derivative = 1;
	layers[1]->HostData[1].Self.Derivative = 0.1;

	weights[0]->HostData[0].Value = 1;
	weights[0]->HostData[1].Value = 0.5;
	weights[0]->HostData[2].Value = 0;
	weights[0]->HostData[3].Value = 1;

//	cout << "Generated random values\n";

	layers[0]->CopyToDevice();
	weights[0]->CopyToDevice();
	layers[1]->CopyToDevice();

	cout << "Copy to device calls after initiated\n";

	ForwardPass<<<N / M, M>>>(
			layers[0]->DeviceData,
			layers[0]->Length,
			weights[0]->DeviceData,
			layers[1]->DeviceData,
			layers[1]->Length);

	BackwardPass<<<N / M, M>>>(
			layers[0]->DeviceData,
			layers[0]->Length,
			weights[0]->DeviceData,
			layers[1]->DeviceData,
			layers[1]->Length);

	cout << "RunPass initiated\n";
//    hipDeviceSynchronize();

	layers[0]->CopyToHost();
	weights[0]->CopyToHost();
	layers[1]->CopyToHost();

	cout << "CopyToHost initiated\n";

    hipDeviceSynchronize();

	cout << "hipDeviceSynchronize finished\n";

    getLastCudaError("Device kernel execution failed.\n");

    cout << "Execution finished, will print\n";

	for(int y = 0; y < N; y++)
	{
		for(int x = 0; x < LAYERS; x++)
		{
			layers[x]->HostData[y].Print();
			cout << " ";

			int nextLayer = x + 1;
			if(nextLayer < LAYERS)
			{
				int block = layers[nextLayer]->Length;
				for(int w = 0; w < block; w++)
				{
					int index = y * block + w;

					cout << y << "->" << w;

					weights[x]->HostData[index].Print();

					cout << " ";
				}
			}
		}

		cout << "\n";
	}

    cout << "print finished\n";

	for(int i = 0; i < layers.size(); i++)
	{
		delete layers[i];
	}

	for(int i = 0; i < weights.size(); i++)
	{
		delete weights[i];
	}

//	delete layers[1];
//	delete weight0to1;
//	delete layer1;

    cout << "dispose finished\n";

    hipDeviceReset();

    cout << "hipDeviceReset finished\n";

    cout << "will exit\n";

//    exit(EXIT_SUCCESS);

	return 0;
}

inline double randomValue()
{
	return 0.5 + (double)rand() / RAND_MAX;
}

void randomValues(double* a, int n)
{
	int i;
	for (i = 0; i < n; ++i)
	{
		a[i] = randomValue();

//		cout << "random = " << a[i] << "\n";
	}
}

void randomValues(Node* a, int n)
{
	int i;
	for (i = 0; i < n; ++i)
	{
		a[i].Self.Value = randomValue();
		a[i].Bias.Value = randomValue();
	}
}

void randomValues(Element* a, int n)
{
	int i;
	for (i = 0; i < n; ++i)
	{
		a[i].Value = randomValue();
	}
}

