#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <array>
using namespace std;

#include "NeuralNetwork.cu"
__global__ void ForwardPass(
		Node* left,
		int leftLength,
		Element* weights,
		Node* right,
		int rightLength)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	double value = 0;

	for(int i = 0; i < leftLength; i++)
	{
		int w = index + i * rightLength;

		value += left[i].Self.Value * weights[w].Value;
	}

	right[index].Self.Value = value;
}

__global__ void BackwardPass(
		Node* left,
		int leftLength,
		Element* weights,
		Node* right,
		int rightLength)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	double total = 0;

	for(int i = 0; i < rightLength; i++)
	{
		int w = index * rightLength + i;

		weights[w].Derivative = weights[w].Value * right[i].Self.Derivative;

		total += weights[w].Derivative;
	}

	left[index].Self.Derivative = total;
}

int main(int argc, char **argv)
{
//	printf ("N = %d \n", N);

	NeuralNetwork* n = new NeuralNetwork();

//	network->

	n->layers[0]->HostData[0].Self.Value = 1;
	n->layers[0]->HostData[1].Self.Value = 2;

	n->layers[1]->HostData[0].Self.Derivative = 1;
	n->layers[1]->HostData[1].Self.Derivative = 0.1;

	n->weights[0]->HostData[0].Value = 1;
	n->weights[0]->HostData[1].Value = 0.5;
	n->weights[0]->HostData[2].Value = 0;
	n->weights[0]->HostData[3].Value = 1;

//	cout << "Generated random values\n";
	n->CopyToDevice();

//	layers[0]->CopyToDevice();
//	weights[0]->CopyToDevice();
//	layers[1]->CopyToDevice();

	cout << "Copy to device calls after initiated\n";


	ForwardPass<<<N / M, M>>>(
			n->layers[0]->DeviceData,
			n->layers[0]->Length,
			n->weights[0]->DeviceData,
			n->layers[1]->DeviceData,
			n->layers[1]->Length);

	BackwardPass<<<N / M, M>>>(
			n->layers[0]->DeviceData,
			n->layers[0]->Length,
			n->weights[0]->DeviceData,
			n->layers[1]->DeviceData,
			n->layers[1]->Length);
//	n->Forward();
//	n->Backward();

	cout << "RunPass initiated\n";
//    hipDeviceSynchronize();
	n->CopyToHost();
//
//	layers[0]->CopyToHost();
//	weights[0]->CopyToHost();
//	layers[1]->CopyToHost();

	cout << "CopyToHost initiated\n";

    hipDeviceSynchronize();

	cout << "hipDeviceSynchronize finished\n";

    getLastCudaError("Device kernel execution failed.\n");

    cout << "Execution finished, will print\n";

    n->Print();
    cout << "print finished\n";

    delete n;

//	delete layers[1];
//	delete weight0to1;
//	delete layer1;

    cout << "dispose finished\n";

    hipDeviceReset();

    cout << "hipDeviceReset finished\n";

    cout << "will exit\n";

//    exit(EXIT_SUCCESS);

	return 0;
}


