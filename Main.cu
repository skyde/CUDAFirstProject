#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <array>
#include "NeuralNetwork.cu"
using namespace std;
#include <vector>
#include <fstream>
#include "ReadMNIST.cpp"
#include "MNISTData.cu"

//int main()
//{
//  vector<vector<double>> ar;
//  ReadMNIST(10000,784,ar);
//
//  return 0;
//}

__global__ void ForwardPass(
		Node* left,
		int leftLength,
		Element* weights,
		Node* right,
		int rightLength)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	double value = 0;

	for(int i = 0; i < leftLength; i++)
	{
		int w = index + i * rightLength;

		value += left[i].Self.Value * weights[w].Value;
	}

	value += right[index].Bias.Value;

	if(right[index].Activation == ActivationTanH)
	{
		value = tanh(value);
	}

	right[index].Self.Value = value;
}

__global__ void BackwardPass(
		Node* left,
		int leftLength,
		Element* weights,
		Node* right,
		int rightLength)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	double value = 0;

	double leftValue = left[index].Self.Value;

	for(int i = 0; i < rightLength; i++)
	{
		int w = index * rightLength + i;
		double rightDerivative = right[i].Self.Derivative;

		weights[w].Derivative = leftValue * rightDerivative;//weights[w].Value * right[i].Self.Derivative;

		value += weights[w].Value * rightDerivative;
	}

	left[index].Bias.Derivative = value;

	if(left[index].Activation == ActivationTanH)
	{
		value = 2.0 / (1.0 + pow(2.718281828459, -2.0 * value)) - 1.0;//tanh(value);
	}

	left[index].Self.Derivative = value;
}

__global__ void CaculateDerivativesFromDifferencePass(
		Node* values,
		double* targets)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	double v = targets[index] - values[index].Self.Value;

	values[index].Self.Derivative = v * STEP_SIZE;
}

__global__ void IterateWeightDerivativePass(Element* weights)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	weights[index].Value += weights[index].Derivative;
}

__global__ void IterateNodeDerivativePass(Node* nodes)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

//	nodes[index].Self.Value += nodes[index].Self.Derivative;
	nodes[index].Bias.Value += nodes[index].Bias.Derivative;
}

void Forward(NeuralNetwork* n)
{
	for(int i = 0; i < n->Layers.size() - 1; ++i)
	{
		int leftLength = n->Layers[i]->Length;
		int rightLength = n->Layers[i + 1]->Length;

//		cout << "\n" << leftLength << "\n";

		ForwardPass<<<rightLength / M, M>>>(
				n->Layers[i]->DeviceData,
				leftLength,
				n->Weights[i]->DeviceData,
				n->Layers[i + 1]->DeviceData,
				rightLength);
	}
}

void CaculateDerivativesFromDifference(NeuralNetwork* n, SharedData<double>* targetValues)
{
	SharedData<Node>* layer = n->Layers[n->Layers.size() - 1];
	int length = layer->Length;

	CaculateDerivativesFromDifferencePass<<<length, 1>>>(
			layer->DeviceData,
			targetValues->DeviceData);
}

void Backward(NeuralNetwork* n)
{
	for(int i = n->Layers.size() - 2; i >= 0; --i)
	{
		int leftLength = n->Layers[i]->Length;
		int rightLength = n->Layers[i + 1]->Length;

		BackwardPass<<<leftLength / M, M>>>(
				n->Layers[i]->DeviceData,
				leftLength,
				n->Weights[i]->DeviceData,
				n->Layers[i + 1]->DeviceData,
				rightLength);
	}
}

void IterateDerivative(NeuralNetwork* n)
{
	for(int i = 0; i < n->Weights.size(); ++i)
	{
		int length = n->Weights[i]->Length;
		IterateWeightDerivativePass<<<length, 1>>>(
				n->Weights[i]->DeviceData);
	}

	// Don't iterate input or output layer
	for(int i = 1; i < n->Layers.size() - 1; ++i)
	{
		int length = n->Layers[i]->Length;
		IterateNodeDerivativePass<<<length, 1>>>(
				n->Layers[i]->DeviceData);
	}
}



int main(int argc, char **argv)
{
	cout << "Reading in MNIST\n";

//	array<double*, MNIST_ELEMENTS_TO_LOAD> values = ReadMNISTData("data0.txt");

	MNISTData* data = new MNISTData(0);
//	vector< vector<double> > ar;
//	ReadMNIST(10000, 784, ar);

	cout << "End reading in MNIST\n";

//	return 0;

	NeuralNetwork* n = new NeuralNetwork();

	n->Layers[0]->HostData[0].Self.Value = 1;
	n->Layers[0]->HostData[1].Self.Value = 5;
	n->Layers[0]->HostData[2].Self.Value = -5;

	// Middle layers are activated
	for(int i = 1; i < LAYERS - 1; ++i)
	{
		n->Layers[1]->HostData[i].Activation = ActivationTanH;
	}

	n->CopyToDevice();

//	double targetValues[2] = { 4.0, -2.0 };

//	array<int>* test;
//
//	cout << test.size();

	SharedData<double>* targetValues = new SharedData<double>(NodesInLayer(LAYERS - 1));
	targetValues->HostData[0] = 12;
	targetValues->HostData[1] = -10;
	targetValues->HostData[2] = 5;
	targetValues->CopyToDevice();

//	cout << "Copy to device calls after initiated\n";

	cout << "\n";

	for(int i = 0; i < 1000; ++i)
	{
		cout << "Epoch " << (i + 1);

		Forward(n);
		CaculateDerivativesFromDifference(n, targetValues);
		Backward(n);
		IterateDerivative(n);

		if(PRINT_ERROR)
		{
			n->CopyToHost();
		}

	    hipDeviceSynchronize();

	    if(PRINT_ERROR)
	    {
			cout << ", error " << n->CaculateError(targetValues->HostData, false);

			cout << " ";

			n->CaculateError(targetValues->HostData, true);
	    }

		cout << "\n";

		if(PRINT_VERBOSE)
		{
		    n->PrintVerbose();
		}
	}


//	#if !PRINT_VERBOSE
//    n->PrintVerbose();
//	#endif

//	cout << "RunPass initiated\n";


//	cout << "CopyToHost initiated\n";

//    hipDeviceSynchronize();

//	cout << "hipDeviceSynchronize finished\n";

//    getLastCudaError("Device kernel execution failed.\n");

//    cout << "Execution finished, will print\n";

//    cout << "print finished\n";

    delete n;
    delete targetValues;

    cout << "dispose finished\n";

    hipDeviceReset();

    cout << "hipDeviceReset finished\n";

    cout << "will exit\n";

//    exit(EXIT_SUCCESS);

	return 0;
}


