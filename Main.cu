#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <array>
using namespace std;

#include "NeuralNetwork.cu"
__global__ void ForwardPass(
		Node* left,
		int leftLength,
		Element* weights,
		Node* right,
		int rightLength)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	double value = 0;

	for(int i = 0; i < leftLength; i++)
	{
		int w = index + i * rightLength;

		value += left[i].Self.Value * weights[w].Value;
	}

	right[index].Self.Value = value;
}

__global__ void BackwardPass(
		Node* left,
		int leftLength,
		Element* weights,
		Node* right,
		int rightLength)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	double total = 0;

	for(int i = 0; i < rightLength; i++)
	{
		int w = index * rightLength + i;

		weights[w].Derivative = weights[w].Value * right[i].Self.Derivative;

		total += weights[w].Derivative;
	}

	left[index].Self.Derivative = total;
}

void Forward(NeuralNetwork* n)
{
	ForwardPass<<<N / M, M>>>(
			n->Layers[0]->DeviceData,
			n->Layers[0]->Length,
			n->Weights[0]->DeviceData,
			n->Layers[1]->DeviceData,
			n->Layers[1]->Length);
}
void Backward(NeuralNetwork* n)
{
	BackwardPass<<<N / M, M>>>(
			n->Layers[0]->DeviceData,
			n->Layers[0]->Length,
			n->Weights[0]->DeviceData,
			n->Layers[1]->DeviceData,
			n->Layers[1]->Length);
}

int main(int argc, char **argv)
{
	NeuralNetwork* n = new NeuralNetwork();

	n->Layers[0]->HostData[0].Self.Value = 1;
	n->Layers[0]->HostData[1].Self.Value = 2;

	n->Layers[1]->HostData[0].Self.Derivative = 0.01;
	n->Layers[1]->HostData[1].Self.Derivative = 0.001;

	n->Weights[0]->HostData[0].Value = 1;
	n->Weights[0]->HostData[1].Value = 0.5;
	n->Weights[0]->HostData[2].Value = 0;
	n->Weights[0]->HostData[3].Value = 1;

	n->CopyToDevice();

//	cout << "Copy to device calls after initiated\n";

	for(int i = 0; i < 10; ++i)
	{
		cout << "\n";
		cout << "Epoch " << i;
		cout << "\n";

		Forward(n);
		Backward(n);

		n->CopyToHost();

	    hipDeviceSynchronize();

	    n->Print();
	}

//	cout << "RunPass initiated\n";


//	cout << "CopyToHost initiated\n";

//    hipDeviceSynchronize();

//	cout << "hipDeviceSynchronize finished\n";

//    getLastCudaError("Device kernel execution failed.\n");

//    cout << "Execution finished, will print\n";

//    cout << "print finished\n";

    delete n;

    cout << "dispose finished\n";

    hipDeviceReset();

    cout << "hipDeviceReset finished\n";

    cout << "will exit\n";

//    exit(EXIT_SUCCESS);

	return 0;
}


