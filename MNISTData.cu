#pragma once
#include <iostream>
using namespace std;
#include "hip/hip_runtime_api.h"
#include "Globals.h"
#include "ReadMNIST.cu"
//#include "SharedData.cu"

class MNISTData
{
public:
	MNISTData(int index)
	{
		string fileName = "data" + to_string(index) + ".txt";

		cout << fileName << "\n";

		for(int i = 0; i < Elements.size(); ++i)
		{
			Elements[i] = new MNISTElement();
		}

		cout << "Start ReadMNISTData\n";
		ReadMNISTData(fileName, Elements);
		cout << "End ReadMNISTData\n";
	}

	virtual ~MNISTData()
	{

	}

//	array<SharedData<double>*>, MNIST_ELEMENTS_TO_LOAD> Values;

	array<MNISTElement*, MNIST_ELEMENTS_TO_LOAD> Elements;

//	SharedData<double>* Values = new SharedData<double>(MNIST_ELEMENT_SIZE);


//	void Dispose()
//	{
//	}

//	void CopyToDevice()
//	{
//		checkCudaErrors(hipMemcpy(DeviceData, HostData, TotalBytes, hipMemcpyHostToDevice));
//	}
//
//	void CopyToHost()
//	{
//		checkCudaErrors(hipMemcpy(HostData, DeviceData, TotalBytes, hipMemcpyDeviceToHost));
//	}
//
//	int TotalBytes;
//	int Length;
//
//	T *HostData;
//	T *DeviceData;
};
