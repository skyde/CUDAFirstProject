#pragma once

#include <iostream>
using namespace std;
#include "hip/hip_runtime_api.h"
#include "Globals.h"
#include "ReadMNIST.cpp"
#include "SharedData.cu"

class MNISTElement
{
	public:
	MNISTElement(double* values)
	{
		Values = new SharedData<double>(MNIST_ELEMENT_SIZE);

//		cout << "Create element\n";

		for(int i = 0; i < MNIST_ELEMENT_SIZE; ++i)
		{
			Values->HostData[i] = values[i];
		}

//		Values->HostData[2] = 5;
		Values->CopyToDevice();
	}

	SharedData<double>* Values;

	virtual ~MNISTElement()
	{
//		Values->
	}
};

class MNISTData
{
public:
	MNISTData(int index)
	{
		string fileName = "data" + to_string(index) + ".txt";

		cout << fileName << "\n";

		array<double*, MNIST_ELEMENTS_TO_LOAD> elements = ReadMNISTData(fileName);

		for(int i = 0; i < elements.size(); ++i)
		{
//			MNISTElement e = ;

			Elements[i] = new MNISTElement(elements[i]);
		}
	}

	virtual ~MNISTData()
	{

	}

//	array<SharedData<double>*>, MNIST_ELEMENTS_TO_LOAD> Values;

	array<MNISTElement*, MNIST_ELEMENT_SIZE> Elements;

//	SharedData<double>* Values = new SharedData<double>(MNIST_ELEMENT_SIZE);


//	void Dispose()
//	{
//	}

//	void CopyToDevice()
//	{
//		checkCudaErrors(hipMemcpy(DeviceData, HostData, TotalBytes, hipMemcpyHostToDevice));
//	}
//
//	void CopyToHost()
//	{
//		checkCudaErrors(hipMemcpy(HostData, DeviceData, TotalBytes, hipMemcpyDeviceToHost));
//	}
//
//	int TotalBytes;
//	int Length;
//
//	T *HostData;
//	T *DeviceData;
};
